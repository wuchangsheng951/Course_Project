
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void vecAdd(int n, float *a, float *b, float *c)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i+=stride)
    c[i] = a[i] + b[i];
}

__global__ 
void init(int n, float *a, float *b) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
  a[i] = 1.0f;
  b[i] = 2.0f;
  }
  }

int main(void)
{
  int N = 1<<20;
  float *x, *y, *z;

  float msec;
  hipEvent_t start, stop;
  int blockSize = 256;
  int numBlocks = 12; // good enough for P620 
  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
  hipMallocManaged(&z, N*sizeof(float));

  // // initialize x and y arrays on the host
  // for (int i = 0; i < N; i++) {
  //   x[i] = 1.0f;
  //   y[i] = 2.0f;
  // }
  init<<<numBlocks, blockSize>>>(N, x, y);

  hipEventCreate(&start);
  hipEventCreate(&stop);


  hipEventRecord(start);
  vecAdd<<<numBlocks, blockSize>>>(N, x, y, z);
  hipEventRecord(stop);
  // Wait for GPU to finish before accessing on host
  hipEventSynchronize(stop);  

  hipEventElapsedTime(&msec, start, stop);
  printf("Kernel time: %f ms\n", msec);
 
  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(z[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  hipFree(z);
  
  return 0;
}
